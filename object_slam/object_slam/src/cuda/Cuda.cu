#include <hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace semantic_slam
{
  namespace cuda
  {
    void deviceSynchronize()
    {
      checkCudaErrors(hipDeviceSynchronize());
    }
  }
}
